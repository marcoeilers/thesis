#include "hip/hip_runtime.h"
#include <part_sort.cuh>
#include <sys/time.h>
#include <scatterKernel.cuh>

#define BIN_BITS 26
#define BITS 0
#define EXP 26
//#define START 0
#define START (BIN_BITS - BITS)
#define BUCKETS (1 << BIN_BITS)

void printDeviceArrays(int *d_array1, int *d_array2, int length)
{
	int *array1 = (int*) malloc(length * sizeof(int));
	int *array2 = (int*) malloc(length * sizeof(int));

	hipMemcpy(array1, d_array1, length * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(array2, d_array2, length * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < length; i++)
	{
		printf("index %i first array %u second array %u\n", i, array1[i], array2[i]);
	}

	free(array2);
	free(array1);
}


void cpuMR(int *keys, int *values, int *result, int num_elements)
{
	for (int i = 0; i < BUCKETS; i++)
		result[i] = 0;

	for (int i = 0; i < num_elements; i++)
		result[keys[i]] += values[i]; 
}

template<int START_BIT, int NO_BITS>
std::pair<timeval, timeval> sortMR(int *h_keys, int *h_values, int *h_result, int num_elements)
{
	typedef int KeyType;
	typedef int ValueType;
	
	// Allocate device data. (We will let the sorting enactor create
	// the "pong" storage if/when necessary.)
	KeyType *d_keys;
	ValueType *d_values;
	int *d_result;
	hipMalloc((void**) &d_keys, sizeof(KeyType) * num_elements);
	hipMalloc((void**) &d_values, sizeof(ValueType) * num_elements);

	// Copy host data to device data
	hipMemcpy(d_keys, h_keys, sizeof(KeyType) * num_elements, hipMemcpyHostToDevice);
	hipMemcpy(d_values, h_values, sizeof(ValueType) * num_elements, hipMemcpyHostToDevice);

	timeval before, between, after;
	
	gettimeofday(&before, NULL);



//++++++ doSortDevice

	// Create a reusable sorting enactor
	b40c::radix_sort::Enactor enactor;

	// Create ping-pong storage wrapper
	b40c::util::DoubleBuffer<KeyType, ValueType> double_buffer;

	// The current key buffer (double_buffer.d_keys[double_buffer.selector]) backs the keys.
	double_buffer.d_keys[double_buffer.selector] = d_keys;
	double_buffer.d_values[double_buffer.selector] = d_values;

// Allocate pong buffer
        int *d_double_keys, *d_double_values;

        hipMalloc((void**) &d_double_keys, sizeof(KeyType) * num_elements);
        hipMalloc((void**) &d_double_values, sizeof(ValueType) * num_elements);
        double_buffer.d_keys[double_buffer.selector ^ 1] = d_double_keys;
        double_buffer.d_values[double_buffer.selector ^ 1] = d_double_values;


	// Sort
//	enactor.Sort(double_buffer, num_elements);
	enactor.OneRunSort<START_BIT, NO_BITS>(double_buffer, num_elements);


	hipDeviceSynchronize();
//printf("error after sort %i\n", hipGetLastError());

        gettimeofday(&between, NULL);

//	printf("\n\n\n\nkeys values after sort\n");
//	printDeviceArrays(d_keys, double_buffer.d_values[double_buffer.selector], 50);	

        hipMalloc((void**) &d_result, sizeof(int) * BUCKETS);
	hipMemset(d_result, 0, sizeof(int) * BUCKETS);

//printf("error after malloc %i\n", hipGetLastError());
        multiReduce(double_buffer.d_keys[double_buffer.selector], double_buffer.d_values[double_buffer.selector], d_result, num_elements);
//printf("error after scatter %i\n", hipGetLastError());
        hipDeviceSynchronize();
        gettimeofday(&after, NULL);

	// Cleanup "pong" storage
	if (d_double_keys) {
		hipFree(d_double_keys);
	}
	if (d_double_values) {
		hipFree(d_double_values);
	}

//	printf("\n\n\n\nvalues after multiReduce\n");
//	printDeviceArrays(d_result, d_result, 100);

	hipMemcpy(h_result, d_result, sizeof(int) * BUCKETS, hipMemcpyDeviceToHost);


	hipFree(d_keys);
	hipFree(d_values);
	hipFree(d_result);

	timeval result1, result2;
	result2.tv_sec = after.tv_sec - between.tv_sec;
	result2.tv_usec = after.tv_usec - between.tv_usec;
	result1.tv_sec = between.tv_sec - before.tv_sec;
	result1.tv_usec = between.tv_usec - before.tv_usec;
	std::pair<timeval, timeval> result(result1, result2);
	return result;
}

int main()
{
	printf("rand max is %i\n", RAND_MAX);
	typedef int KeyType;
	typedef int ValueType;

	unsigned int num_elements = 1 << EXP;


        // Allocate host problem data
	KeyType *h_keys = new KeyType[num_elements];
	ValueType *h_values = new ValueType[num_elements];
	int *h_result = new int[BUCKETS];
	int *result_cpu = new int[BUCKETS];

        // Initialize host problem data
	int index = rand() % BUCKETS;
        for (int i = 0; i < num_elements; ++i)
        {
		h_keys[i] = index;
//                h_keys[i] = rand() % BUCKETS;
                h_values[i] = rand() % 100;
// 		if (i < 50)
//		printf("original key value %i %i %i\n", i, h_keys[i], h_values[i]);
        }
	std::pair<timeval, timeval> result;
	result = sortMR<START, BITS>(h_keys, h_values, h_result, num_elements);
	
	cpuMR(h_keys, h_values, result_cpu, num_elements);


	bool correct = true;
	for (int i = 0; i < BUCKETS; i++)
	{
		if (h_result[i] != result_cpu[i])
		{
			printf("i sense a discrepancy! %i %i %i\n", i, h_result[i], result_cpu[i]);
			correct = false;
		}
	}
	if (correct)
	printf("correct result!\n");

	printf("last error is %i\n", hipGetLastError());

	float time1 = result.first.tv_sec * 1e9 + result.first.tv_usec * 1e3;
	float time2 = result.second.tv_sec * 1e9 + result.second.tv_usec * 1e3;
	printf("%i\t%i\t%f\t%f\t%f\n", num_elements, BITS, time1 / num_elements, time2 / num_elements, (time1 + time2) / num_elements);

	delete(result_cpu);
	delete(h_keys);
        delete(h_values);
	delete(h_result);
	return 0;
}
