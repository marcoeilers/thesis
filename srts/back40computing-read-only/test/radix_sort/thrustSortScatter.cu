#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
//#include <thrust/sort.h>
//#include <thrust/scatter.h>
//#include <thrust/system_error.h>
//#include <thrust/device_ptr.h>
#include "part_sort.cuh"

#define ALLOC_SIZE 1000000000

#define SIZE_NEED 1

int main() 
{
hipError_t error;
hipDevice_t device;
hipCtx_t context;

  size_t freemem;
  size_t totalmem;

hipInit(0); // Initialize CUDA
    hipDeviceGet( &device, 0 ); // Get handle for device
    hipCtxCreate( &context, 0, device ); // Create context

  hipMemGetInfo(&freemem, &totalmem);
  printf("free %lu total %lu\n", freemem, totalmem);

int *d_zero;
error = hipMalloc((void**) &d_zero, ALLOC_SIZE); 
if (error != hipSuccess)
    printf("error during initial cudamalloc\n");

int size = 1 << 20;

  while (size * SIZE_NEED * sizeof(int) < ALLOC_SIZE)
    {
      
      int *keys = (int*) malloc(size * sizeof(int));
      int *values = (int*) malloc(size * sizeof(int));
//      int *result = (int*) malloc(size * sizeof(int));

      for (int i = 0; i < size; i++)
	{
	  keys[i] = rand() % size;
	  values[i] = rand();
	}
      
      
      
//      int *d_keys, *d_values, *d_result;
/*
      if (hipSuccess != hipMalloc((void**) &d_keys, size * sizeof(int)))
        printf("error hipMalloc %i", hipGetLastError());
      if (hipSuccess != hipMalloc((void**) &d_values, size * sizeof(int)))
        printf("error hipMalloc");
      if (hipSuccess != hipMalloc((void**) &d_result, size * sizeof(int)))
	printf("error hipMalloc");
*/

//      d_keys = d_zero;
//      d_values = d_keys + size;
//      d_result = d_values + size;

//      hipDeviceSynchronize(); 
//      thrust::device_ptr<int> keys_ptr(d_keys );
//      thrust::device_ptr<int> values_ptr(d_values );
//      thrust::device_ptr<int> result_ptr(d_result);

//      timeval before, after;
//      gettimeofday(&before, NULL);
/*
      error = hipMemcpy(d_values, values, sizeof(int) * size, hipMemcpyHostToDevice);
      if (error != hipSuccess)
          printf("error copy values\n");

      error = hipMemcpy(d_keys, keys, sizeof(int) * size, hipMemcpyHostToDevice);
      if (error != hipSuccess)
          printf("error during copy keys\n");
*/     
//      thrust::stable_sort_by_key(keys_ptr, keys_ptr + size, values_ptr);

//      thrust::scatter(values_ptr, values_ptr + size, keys_ptr, result_ptr);
//      thrust::gather(keys_ptr, keys_ptr + size, values_ptr, result_ptr);
      timeval result;
      result = one_run_sort<0, 16>(keys, values, size, false);

//      hipDeviceSynchronize();
//      gettimeofday(&after, NULL);

      float time;
      time = result.tv_sec * 1e9 + result.tv_usec * 1e3;
     
      printf("%i\t%f\n", size, time / size);
      
    
      free(keys);
      free(values);
//      free(result);
      size *= 2;
    }
hipFree(&d_zero);
}


