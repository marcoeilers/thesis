#include <b40c/radix_sort/enactor.cuh>
#include <b40c/util/multi_buffer.cuh>

#include <sys/time.h>
#include <scatterKernel.cuh>
#include <assert.h>
#include <thrust/scan.h>
#include "valIndex.h"


#define CHECK 128
#define BINS_EXP 26
#define ELEMENTS_EXP 26

void multiScanCpu(int *keys, valIndex *values, int *result, int num_elements)
{
	int *buckets = (int*) malloc((1 << BINS_EXP) *sizeof(int));
	for (int i = 0; i < (1 << BINS_EXP); i++)
		buckets[i] = 0;

	for (int i = 0; i < num_elements; i++)
	{
		buckets[keys[i]] += values[i].value;
		result[i] = buckets[keys[i]];
//                buckets[keys[i]] += values[i].value;
	}
	free(buckets);
}  

template<typename ValueType, int START_BIT, int NO_BITS>
std::pair<timeval, timeval> multiScan(int *h_keys, ValueType *h_values, int *h_result, int num_elements, bool keys_only)
{
	typedef int KeyType;
	
	// Allocate device data. (We will let the sorting enactor create
	// the "pong" storage if/when necessary.)
	KeyType *d_keys;
	ValueType *d_values;
	int *d_result;
	hipMalloc((void**) &d_keys, sizeof(KeyType) * num_elements);
	hipMalloc((void**) &d_values, sizeof(ValueType) * num_elements);

	// Copy host data to device data
	hipMemcpy(d_keys, h_keys, sizeof(KeyType) * num_elements, hipMemcpyHostToDevice);
	hipMemcpy(d_values, h_values, sizeof(ValueType) * num_elements, hipMemcpyHostToDevice);

	timeval before, between, after;
	
	gettimeofday(&before, NULL);



	// Create a reusable sorting enactor
	b40c::radix_sort::Enactor enactor;

	// Create ping-pong storage wrapper
	b40c::util::DoubleBuffer<KeyType, ValueType> double_buffer;

	// The current key buffer (double_buffer.d_keys[double_buffer.selector]) backs the keys.
	double_buffer.d_keys[double_buffer.selector] = d_keys;
	double_buffer.d_values[double_buffer.selector] = d_values;

	// Allocate pong buffer
        int *d_double_keys;
	double *d_double_values;

        hipMalloc((void**) &d_double_keys, sizeof(KeyType) * num_elements);
        hipMalloc((void**) &d_double_values, sizeof(ValueType) * num_elements);
        double_buffer.d_keys[double_buffer.selector ^ 1] = d_double_keys;
        double_buffer.d_values[double_buffer.selector ^ 1] = d_double_values;


	// Sort
//	enactor.Sort(double_buffer, num_elements);
	enactor.OneRunSort<START_BIT, NO_BITS>(double_buffer, num_elements);

	hipDeviceSynchronize();
//printf("error after sort %i\n", hipGetLastError());
	
//	valIndex *checkValues = (valIndex*)malloc(sizeof(valIndex) * CHECK);
//	int *checkKeys = (int*)malloc(sizeof(int) * CHECK);
/*	hipMemcpy(checkValues, double_buffer.d_values[double_buffer.selector], sizeof(valIndex) * CHECK, hipMemcpyDeviceToHost);
	hipMemcpy(checkKeys, d_keys, sizeof(int) * CHECK, hipMemcpyDeviceToHost);

	for (int i = 0; i < CHECK - 1; i++)
	{
	if (!(checkKeys[i] <= checkKeys[i+1]))
		printf("keys not ordered\n");
	int origIndex = checkValues[i].index;
	if (h_keys[origIndex] != checkKeys[i])
		printf("not the right index\n");
	if (((valIndex*)(h_values))[origIndex].value != checkValues[i].value)
		printf("not the right value!\n");
	printf("%i\t%i\t%i\n", checkKeys[i], checkValues[i].index, checkValues[i].value);

	}
printf("\n\n\n\n");
*/
	gettimeofday(&between, NULL);
	thrust::equal_to<int> binary_pred;
	valIndexAdd     binary_op;
	thrust::device_ptr<int> keys_ptr(double_buffer.d_keys[double_buffer.selector]);
	thrust::device_ptr<valIndex> values_ptr((valIndex*)double_buffer.d_values[double_buffer.selector]);

	thrust::inclusive_scan_by_key(keys_ptr, keys_ptr + num_elements, values_ptr, values_ptr, binary_pred, binary_op);


/*	hipMemcpy(checkValues, double_buffer.d_values[double_buffer.selector], sizeof(valIndex) * CHECK, hipMemcpyDeviceToHost);
        hipMemcpy(checkKeys, d_keys, sizeof(int) * CHECK, hipMemcpyDeviceToHost);
        for (int i = 0; i < CHECK - 1; i++)
        {
        if (!(checkKeys[i] <= checkKeys[i+1]))
                printf("keys not ordered\n");
        printf("%i\t%i\t%i\n", checkKeys[i], checkValues[i].index, checkValues[i].value);

        }
*/
	

        hipMalloc((void**) &d_result, sizeof(int) * num_elements);

        scatterValIndex((valIndex*)double_buffer.d_values[double_buffer.selector], d_result, num_elements);

        hipDeviceSynchronize();
        gettimeofday(&after, NULL);

	// Cleanup "pong" storage
	if (d_double_values) {
		hipFree(d_double_values);
	}
	if (d_double_keys) {
		hipFree(d_double_keys);
	}

	hipMemcpy(h_result, d_result, sizeof(int) * num_elements, hipMemcpyDeviceToHost);
/*
for (int i = 0; i < CHECK; i++)
{
        int currentIndex = checkValues[i].index;
        int currentValue = checkValues[i].value;
	if (h_result[currentIndex] != currentValue)
		printf("value not correctly scattered!\n");
	else
		printf("correct value\n");
}
*/


	hipFree(d_keys);
	hipFree(d_values);

	timeval result1, result2;
	result2.tv_sec = after.tv_sec - between.tv_sec;
	result2.tv_usec = after.tv_usec - between.tv_usec;
	result1.tv_sec = between.tv_sec - before.tv_sec;
	result1.tv_usec = between.tv_usec - before.tv_usec;
	std::pair<timeval, timeval> result(result1, result2);
	return result;
}

int main()
{
	typedef int KeyType;
	typedef valIndex ValueType;;
printf("valIndex %i float %i double %i\n", sizeof(valIndex), sizeof(float), sizeof(double));
	unsigned int num_elements = 1 << ELEMENTS_EXP;


        // Allocate host problem data
	KeyType *h_keys = new KeyType[num_elements];
	ValueType *h_values = new ValueType[num_elements];
	int *h_result = new int[num_elements];
	int *h_result_cpu = new int[num_elements];

        // Initialize host problem data
	int index = rand() % (1 << BINS_EXP);
        for (int i = 0; i < num_elements; ++i)
        {
                h_keys[i] = index;
                h_values[i].index = i;
		h_values[i].value = rand() % 1000;
		h_result[i] = 1;
//		printf("%i\t%i\t%i\n", i, h_keys[i], h_values[i].value);
        }
//printf("\n\n\n");
	timeval before, after;
	gettimeofday(&before, NULL);
	multiScanCpu(h_keys, h_values, h_result_cpu, num_elements);
	gettimeofday(&after, NULL);
	std::pair<timeval, timeval> result;
	result = multiScan<double, 0, BINS_EXP>(h_keys, (double*)h_values, h_result, num_elements, false);
	printf("cpu gpu ns/int %f\t%f\n", ((after.tv_sec - before.tv_sec) * 1e9 + (after.tv_usec - before.tv_usec) * 1e3) / num_elements, ((result.first.tv_sec * 1e9 + result.first.tv_usec * 1e3) + (result.second.tv_sec * 1e9 + result.second.tv_usec * 1e3)) / num_elements);
printf("now doing that checking thing...\n");
	for (int i = 0; i < num_elements; i++)
	{
		if(h_result[i] != h_result_cpu[i])
			printf("results unequal: %i %i %i\n", i, h_result[i], h_result_cpu[i]);
	}

	delete(h_keys);
        delete(h_values);
	delete(h_result);
	return 0;
}
