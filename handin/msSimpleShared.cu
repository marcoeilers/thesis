#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include "cudpp.h"

#define BINS_EXP 9
#define BINS (1 << BINS_EXP)
#define BUCKETS BINS
#define NT 32
#define VT 8
#define VT1 (VT + 1)



void cudppMS(int blocks, int *in, int *out, size_t pitch);


__global__ void multiReduceKernel(int *indices, int *values, int *result, int *allBuckets, int segLength, size_t pitch)
{
__shared__ int blockBuckets[NT * BUCKETS];
__shared__ int sIndices[VT1 * NT];
__shared__ int sValues[VT1 * NT];

        int *myBuckets = &blockBuckets[threadIdx.x];

        int blocks = gridDim.x;
        int blockId = blockIdx.x;
        int threadId = threadIdx.x;
//        int threadspb = blockDim.x;
        int hists = blocks * NT;
        int todopb = NT * segLength;

        int *ourIndices = indices + blockId * todopb;
        int *ourValues = values + blockId * todopb;
        int *ourResult = result + blockId * todopb;
        int *myResult = ourResult + threadId * segLength;
        int *myIndices = &sIndices[threadId * VT1];
        int *myValues = &sValues[threadId * VT1];

        // initialize buckets to zero
        for (int i = threadIdx.x; i < NT * BUCKETS; i += NT)
                blockBuckets[i] = 0;

        __syncthreads();

        #pragma unroll 16
        for (int it = 0; it < segLength; it += VT)
        {
        int offset = it;
        int todo = min(segLength - offset, VT);

        for(int i = threadIdx.x; i < todo * NT; i += NT)
        {
                int segm = i / todo;
                int ind = i % todo;
                int sIndex = segm * VT1 + ind;
                int index = segm * segLength + ind + offset;
                sIndices[sIndex] = ourIndices[index];
                sValues[sIndex] = ourValues[index];
        }

        __syncthreads();

        for (int i = 0; i < todo; i++)
        {
                int currentLabel = myIndices[i];
                myResult[offset + i] = myBuckets[currentLabel * NT];
                myBuckets[currentLabel * NT] += myValues[i];
        }

        __syncthreads();

        }

        int *myGlobalBuckets = allBuckets + blockIdx.x * NT + threadId;
	int pitchByInt = pitch / sizeof(int);
        // write results to global memory
        for (int i = 0; i < BUCKETS; i++)
        {
//if (blockIdx.x == 0 && threadIdx.x < 4) printf("thread %i writing result %i for bucket %i\n", threadIdx.x, blockBuckets[threadId + i * NT], i);
                myGlobalBuckets[i * pitchByInt] = blockBuckets[threadId + i * NT];
//                myResult[i * hists] = blockBuckets[threadId + i * NT];
        }
}


/*
__global__ void blockSegScan(int *values, int *labels, int *result, int *allBuckets, size_t pitch)
{
	int tid = threadIdx.x;
	int *ourValues = values + blockIdx.x * WORK_PER_BLOCK;
	labels = labels + blockIdx.x * WORK_PER_BLOCK;
	result = result + blockIdx.x * WORK_PER_BLOCK;
	int pitchByInt = pitch / sizeof(int);
//	for (int i = tid; i < BINS; i += THREADS_PER_BLOCK)
//		allBuckets[i * pitchByInt + blockIdx.x] = 0;

	
//        for (int i = tid; i < pitchByInt * BINS; i += THREADS_PER_BLOCK)
//                allBuckets[blockIdx.x * BINSi] = 0;
//        __syncthreads();

	typedef mgpu::CTASegScan<THREADS_PER_BLOCK, mgpu::plus<int> > SegScan;
	typedef hipcub::BlockRadixSort<int, THREADS_PER_BLOCK, ITEMS_PER_THREAD, IndexValue> BlockRadixSortT;
//	typedef hipcub::BlockRadixSort<int, THREADS_PER_BLOCK, ITEMS_PER_THREAD, IndexValue, 4, 1, hipcub::BLOCK_SCAN_RAKING_MEMOIZE> BlockRadixSortT;

	union Shared {
		typename BlockRadixSortT::TempStorage sort;
		LabelIndexValue ilvs[THREADS_PER_BLOCK * ITEMS_PER_THREAD];
		struct {
			typename SegScan::Storage segScanStorage;
			int lastValue;
			int lastLabel;
		} segScan;
	};
	__shared__ Shared shared;

	int myLabels[ITEMS_PER_THREAD];

	IndexValue myValues[ITEMS_PER_THREAD];

	for (int i = 0; i < ITEMS_PER_THREAD; i++)
	{
		myLabels[i] = labels[ITEMS_PER_THREAD * tid + i];
		myValues[i].index = ITEMS_PER_THREAD * tid + i;
		myValues[i].value = ourValues[ITEMS_PER_THREAD * tid + i];
	}
	__syncthreads();
	BlockRadixSortT(shared.sort).Sort(myLabels, myValues, 0, BINS_EXP);

	__syncthreads();

	for (int i = 0; i < ITEMS_PER_THREAD; i++)
	{
		shared.ilvs[tid * ITEMS_PER_THREAD + i].index = myValues[i].index;
		shared.ilvs[tid * ITEMS_PER_THREAD + i].label = myLabels[i];
		shared.ilvs[tid * ITEMS_PER_THREAD + i].value = myValues[i].value;
	}
	__syncthreads();

	int myLabelsPred[ITEMS_PER_THREAD];
	int myFlags[ITEMS_PER_THREAD];
	for (int i = 0; i < ITEMS_PER_THREAD; i++)
	{
		myValues[i].index = shared.ilvs[tid + i * THREADS_PER_BLOCK].index;
		myValues[i].value = shared.ilvs[tid + i * THREADS_PER_BLOCK].value;
		myLabels[i] = shared.ilvs[tid + i * THREADS_PER_BLOCK].label;
		myLabelsPred[i] = i ? shared.ilvs[tid - 1 + i * THREADS_PER_BLOCK].label : tid ? shared.ilvs[tid - 1 + i * THREADS_PER_BLOCK].label : 0;

		if (i)
			myFlags[i] = shared.ilvs[tid + i * THREADS_PER_BLOCK - 1].label != shared.ilvs[tid + i * THREADS_PER_BLOCK].label;
		else
			myFlags[i] = tid ? shared.ilvs[tid + i * THREADS_PER_BLOCK - 1].label != shared.ilvs[tid + i * THREADS_PER_BLOCK].label : 1;
	}

	__syncthreads();

	int carryOut;
	for (int i = 0; i < ITEMS_PER_THREAD; i++)
	{
		int x = SegScan::SegScan(tid, myValues[i].value, myFlags[i], shared.segScan.segScanStorage, &carryOut, 0);

		__syncthreads();

		if (myFlags[i] && myLabels[i] != 0)
		{
			allBuckets[((myLabelsPred[i]) * pitchByInt) + blockIdx.x] = (i && myLabelsPred[i] == shared.segScan.lastLabel) ? x + shared.segScan.lastValue : x;
		}

                if (i != 0 && myLabels[i] == shared.segScan.lastLabel)
                {
                        x += shared.segScan.lastValue;
                }

                int writeResult = myFlags[i] ? 0 : x;

		result[myValues[i].index] = writeResult;


		__syncthreads();
		if (threadIdx.x == (THREADS_PER_BLOCK - 1))
		{
                        allBuckets[((myLabels[ITEMS_PER_THREAD - 1]) * pitchByInt) + blockIdx.x] = carryOut;
			shared.segScan.lastValue = carryOut;
			shared.segScan.lastLabel = myLabels[i];
		}
	}

}
*/


/*
__global__
void addRemainder(int *bucketSums, int *indices, int *result, size_t pitch)
{
__shared__ int bins[BINS];
int pitchByInt = pitch / sizeof(int);
int work = WORK_PER_BLOCK;

int blocks = gridDim.x;

#pragma unroll 2
for (int i = threadIdx.x; i < BINS; i+=THREADS_PER_BLOCK)
{
	bins[i] = bucketSums[blockIdx.x + i * pitchByInt];
}
__syncthreads();


int *myIndices = &indices[work * blockIdx.x];
int *myResult = &result[work * blockIdx.x];

#pragma unroll ITEMS_PER_THREAD
for (int i = threadIdx.x; i < work; i+=THREADS_PER_BLOCK)
{
//if (blockIdx.x == 1 && i < 100) printf("adding %i to result %i, label is %i\n", bins[myIndices[i]], i + blockIdx.x * work, myIndices[i]);
	myResult[i] += bins[myIndices[i]];
}

}
*/

__global__
void addRemainderGlobalShared(int *bucketSums, int *indices, int *result, int segLength, size_t pitch)
{
int pitchByInt = pitch / sizeof(int);

int blocks = gridDim.x;
int nThreads = blockDim.x;
__syncthreads();

int *myIndices = &indices[segLength * blockIdx.x];
int *myResult = &result[segLength * blockIdx.x];

#pragma unroll ITEMS_PER_THREAD
for (int i = threadIdx.x; i < segLength; i+= nThreads)
{
        myResult[i] += bucketSums[blockIdx.x + (myIndices[i]) * pitchByInt]; //bins[myIndices[i]];
}

}

/*
void comparePartials(int *d_indices, int *d_values, int *d_results, int *d_allBuckets, int num_elements, size_t pitch)
{
int pitchByInt = pitch / sizeof(int);
int *indices, *values, *results, *allBuckets, *cresults, *callBuckets;
indices = (int*) malloc(num_elements * sizeof(int) * 4);
values = indices + num_elements;
results = values + num_elements;
cresults = results + num_elements;
allBuckets = (int*) malloc(pitch * BINS * 2);
callBuckets = allBuckets + pitchByInt * BINS;

hipMemcpy(indices, d_indices, sizeof(int) * num_elements, hipMemcpyDeviceToHost);
hipMemcpy(values, d_values, sizeof(int) * num_elements, hipMemcpyDeviceToHost);
hipMemcpy(results, d_results, sizeof(int) * num_elements, hipMemcpyDeviceToHost);
hipMemcpy(allBuckets, d_allBuckets, pitch * BINS, hipMemcpyDeviceToHost);

for (int i = 0; i < num_elements; i += NT * VT)
{
	int block = i / (THREADS_PER_BLOCK * ITEMS_PER_THREAD);

	int *curIndices = indices + i;
	int *curValues = values + i;
	int *curResults = cresults + i;

	for (int j = 0; j < BINS; j++)
		callBuckets[j] = 0;
	for (int j = 0; j < THREADS_PER_BLOCK * ITEMS_PER_THREAD; j++)
	{
		curResults[j] = callBuckets[curIndices[j]];
		callBuckets[curIndices[j]] += curValues[j];
	}

	for (int j = 0; j < THREADS_PER_BLOCK * ITEMS_PER_THREAD; j++)
	{
		if (results[i + j] != curResults[j])
			printf("different result block %i in %i should be %i is %i, label %i\n", block, i + j, curResults[j], results[i+j], curIndices[j]);
	}

	for (int j = 0; j < BINS; j++)
	{
		if (callBuckets[j] != allBuckets[j * pitchByInt + block])
			printf("different bucket block %i bin %i is %i should be %i\n", block, j, allBuckets[j * pitchByInt + block], callBuckets[j]);
	}
}

}
*/


void invokeMultiScan(int *indices, int *values, int *results, int num_elements)
{
hipDeviceProp_t props;
hipGetDeviceProperties(&props, 0);

int shmemPerBlock = NT * BUCKETS * sizeof(int) + (2 * NT * VT1 * sizeof(int));
int blocksPerSm = props.sharedMemPerBlock / shmemPerBlock;

int totalBlocks = props.multiProcessorCount * blocksPerSm;
printf("%i SMs, %i bytes shared memory, we use %i bytes per block, therefore have %i blocks and segLEngth %i\n", props.multiProcessorCount, props.sharedMemPerBlock, shmemPerBlock, totalBlocks, num_elements / (NT * totalBlocks));

int hists = NT * totalBlocks;


int segLength = num_elements / hists;

int *allBuckets;
int *allBucketsResult;
size_t pitch;
size_t pitch_result;
hipMallocPitch((void**)&allBuckets, &pitch, hists * sizeof(int),  BINS);
hipMallocPitch((void**)&allBucketsResult, &pitch_result, hists * sizeof(int), BINS);

printf("temp memory allocation error: %i\n", hipGetLastError());

dim3 dimBlock(NT, 1);
dim3 dimGrid(totalBlocks, 1);
hipFuncSetCacheConfig(reinterpret_cast<const void*>(multiReduceKernel), hipFuncCachePreferShared);
hipFuncSetCacheConfig(reinterpret_cast<const void*>(addRemainderGlobalShared), hipFuncCachePreferShared);


timeval before, between1, between2, after;

gettimeofday(&before, NULL);
//hipMemset(results, 0, num_elements * sizeof(int));
hipMemset(allBuckets, 0, pitch * BINS);

//naiveGlobalSeparateMultiScan<<<blocks, THREADS_PER_BLOCK>>>(indices, values, allBuckets, pitch);
multiReduceKernel<<<totalBlocks, NT>>>(indices, values, results, allBuckets, segLength, pitch);

hipDeviceSynchronize();
printf("error after blockSegScan is %i\n", hipGetLastError());
//comparePartials(indices, values, results, allBuckets, num_elements, pitch);

gettimeofday(&between1, NULL);

cudppMS(hists, allBuckets, allBucketsResult, pitch);
hipDeviceSynchronize();

gettimeofday(&between2, NULL);

	addRemainderGlobalShared<<<hists, 128>>>(allBucketsResult, indices, results, segLength, pitch);


hipDeviceSynchronize();
gettimeofday(&after, NULL);

float time3 = (after.tv_sec - between2.tv_sec) * 1e9 + (after.tv_usec - between2.tv_usec) * 1e3;
float time2 = (between2.tv_sec - between1.tv_sec) * 1e9 + (between2.tv_usec - between1.tv_usec) * 1e3;
float time1 = (between1.tv_sec - before.tv_sec) * 1e9 + (between1.tv_usec - before.tv_usec) * 1e3;
printf("%i\t%f\t%f\t%f\t%f\n", BUCKETS, time1 / num_elements, time2 / num_elements, time3 / num_elements, (time1 + time2 + time3) / num_elements);
}


void cudppMS(int blocks, int *in, int *out, size_t pitch)
{
// Initialize the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_INT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
    
    CUDPPHandle scanplan = 0;
    CUDPPResult res = cudppPlan(theCudpp, &scanplan, config, blocks, BINS, pitch / sizeof(int));  
    if (CUDPP_SUCCESS != res)
    {
        printf("Error creating CUDPPPlan\n");
        exit(-1);
    }

// Run the scan
    res = cudppMultiScan(scanplan, out, in, blocks, BINS);
    if (CUDPP_SUCCESS != res)
    {
        printf("Error in cudppScan()\n");
        exit(-1);
    }
}


void multiScanCpu(int *indices, int *values, int *result, int num_elements)
{
int buckets[BINS];

for (int i = 0; i < BINS; i++)
	buckets[i] = 0;

for (int i = 0; i < num_elements; i++)
{
//if (i == 3 * THREADS_PER_BLOCK * ITEMS_PER_THREAD) {for (int j = 0; j < BINS; j++) printf("in block 3 bucket %i should be %i\n", j, buckets[j]);}

	result[i] = buckets[indices[i]];
	buckets[indices[i]] += values[i];
}
}



int main()
{
hipDeviceProp_t props;
hipGetDeviceProperties(&props, 0);

int shmemPerBlock = NT * BUCKETS * sizeof(int) + (2 * NT * VT1 * sizeof(int));
int blocksPerSm = props.sharedMemPerBlock / shmemPerBlock;

int totalBlocks = props.multiProcessorCount * blocksPerSm;
//printf("%i SMs, %i bytes shared memory, we use %i bytes per block, therefore have %i blocks\n", props.multiProcessorCount, props.sharedMemPerBlock, shmemPerBlock, totalBlocks);

int hists = NT * totalBlocks;


int size = ((1 << 26) / hists) * hists;
int *keys = (int*) malloc(size * sizeof(int));
int *values = (int*) malloc(size * sizeof(int));
int *result = (int*) malloc(size * sizeof(int));
int *result_cpu = (int*) malloc(size * sizeof(int));
int index = rand() % BINS; 
int *d_keys, *d_values, *d_results;

{
for (int i = 0; i < size; i++)
{
	values[i] = rand() % 10;
	keys[i] = rand() % BUCKETS;
//if (i < 4 * ITEMS_PER_THREAD)
//	printf("original index %i label %i value %i \n", i, keys[i], values[i]);
	result_cpu[i] = 15;
}


hipMalloc((void**) &d_keys, size * sizeof(int));
hipMalloc((void**) &d_values, size * sizeof(int));
hipMalloc((void**) &d_results, size * sizeof(int));

hipMemcpy(d_keys, keys, size * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_values, values, size * sizeof(int), hipMemcpyHostToDevice);

invokeMultiScan(d_keys, d_values, d_results, size);

hipMemcpy(result, d_results, size * sizeof(int), hipMemcpyDeviceToHost);

timeval before, after;
gettimeofday(&before, NULL);
multiScanCpu(keys, values, result_cpu, size);
gettimeofday(&after, NULL);

printf("cpu needed %f ns/int \n", ((after.tv_sec - before.tv_sec) * 1e9 + (after.tv_usec - before.tv_usec) * 1e3) / size);
int correct = 1;
for (int i = 0; i < size; i++)
{
	if (result[i] != result_cpu[i])
	{
//		printf("I sense a discrepancy! %i %i %i value %i label %i\n", i, result[i], result_cpu[i], values[i], keys[i]);
		correct = 0;
	}
//	else
//		printf("correct!\n");
}
printf("correct? %i\n", correct);
printf("last error: %i\n", hipGetLastError());

hipFree(d_keys);
hipFree(d_values);
hipFree(d_results);
}

{
printf("now deg\n");
for (int i = 0; i < size; i++)
{
        values[i] = rand() % 10;
        keys[i] = index;
//if (i < 4 * ITEMS_PER_THREAD)
//      printf("original index %i label %i value %i \n", i, keys[i], values[i]);
        result_cpu[i] = 15;
}


hipMalloc((void**) &d_keys, size * sizeof(int));
hipMalloc((void**) &d_values, size * sizeof(int));
hipMalloc((void**) &d_results, size * sizeof(int));

hipMemcpy(d_keys, keys, size * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_values, values, size * sizeof(int), hipMemcpyHostToDevice);

invokeMultiScan(d_keys, d_values, d_results, size);

hipMemcpy(result, d_results, size * sizeof(int), hipMemcpyDeviceToHost);

timeval before, after;
gettimeofday(&before, NULL);
multiScanCpu(keys, values, result_cpu, size);
gettimeofday(&after, NULL);

printf("cpu needed %f ns/int \n", ((after.tv_sec - before.tv_sec) * 1e9 + (after.tv_usec - before.tv_usec) * 1e3) / size);
int correct = 1;
for (int i = 0; i < size; i++)
{
        if (result[i] != result_cpu[i])
        {
//              printf("I sense a discrepancy! %i %i %i value %i label %i\n", i, result[i], result_cpu[i], values[i], keys[i]);
                correct = 0;
        }
//      else
//              printf("correct!\n");
}
printf("correct? %i\n", correct);
printf("last error: %i\n", hipGetLastError());

hipFree(d_keys);
hipFree(d_values);
hipFree(d_results);

}
}


